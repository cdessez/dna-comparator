
#include <hip/hip_runtime.h>
extern "C"
__global__ void distance(int nu_g, int nt_g, int nk_g, int *u, int *t, int *d0, int *d1, int *d2)
{
	// allocate local copies of this variable for a faster access
	int nu, nt, nd, nk;
	nu = nu_g; nt = nt_g; nk = nk_g;
	
	// local indexes and temp variables
    int k, l, max_l, tmp_min;
    int *tmp_d;
    
    for(k = 2 ; k <= nk ; k++){
    	tmp_d = d0;
    	d0 = d1;
    	d1 = d2;
    	d2 = tmp_d;
    	l = (k > nu ? (k - nu) : 0) + threadIdx.x;
    	max_l = k > nt ? nt : k;
    	
    	for(; l <= max_l ; l += blockDim.x){
    		if (l == 0){
    			d2[0] = d1[0] + 1;
    		} else if (l == k){
    			d2[l] = d1[l-1] + 1;
    		} else {
    			tmp_min = d1[l-1] > d1[l] ? (d1[l] + 1) : (d1[l-1] + 1);
    			if (u[k-l-1] != t[l-1])
    				d2[l] = tmp_min < (d0[l-1] + 1) ? tmp_min : (d0[l-1] + 1);
    			else
    				d2[l] = tmp_min < d0[l-1] ? tmp_min : d0[l-1];
    		}
    	}
    	
    	__syncthreads();
    }
}
